#include "hip/hip_runtime.h"
#include "cell/mod.hpp"
#include "cuda_info.hpp"
#include "errors.hpp"
#include "kernels/lstm_cell.hpp"
#include "types/layout.hpp"

namespace tiledcuda::kernels {

using namespace tiledcuda::cell;
using namespace tiledcuda::cell::copy;
using namespace tiledcuda::cell::compute;

namespace tl = tiledcuda::cell::tile_layout;

template <typename Element, typename KeTraits>
__global__ void dyn_lstm_gate(const Element* ws, const Element* us,
                              const Element* xs, const Element* hs, Element* ts,
                              const int m, const int n, const int k) {
    extern __shared__ __align__(sizeof(double)) unsigned char shared_buf[];
    auto* shm = reinterpret_cast<Element*>(shared_buf);

    const int kN = n;
    const int kK = k;

    // CTA GEMM shape
    const int kTM = KeTraits::kTM;
    const int kTN = KeTraits::kTN;
    const int kTK = KeTraits::kTK;

    // Advance to the global data tile to the current CTA.
    Element* gxs_ptr = const_cast<Element*>(xs) + blockIdx.y * kK * kTN;
    Element* ghs_ptr = const_cast<Element*>(hs) + blockIdx.y * kK * kTN;
    Element* gws_ptr = const_cast<Element*>(ws) + blockIdx.x * kK * kTM;
    Element* gus_ptr = const_cast<Element*>(us) + blockIdx.x * kK * kTM;

    Element* gts_ptr = ts + blockIdx.x * kTM * kN + blockIdx.y * kTN;

    int total_block_x = gridDim.x;
    int current_block_x = blockIdx.x;

    // pointers to shared memory tiles
    Element* sws_ptr = shm;
    Element* sxs_ptr = shm + kTM * kTK;
    Element* sus_ptr = shm + kTM * kTK + kTK * kTN;
    Element* shs_ptr = shm + kTM * kTK + kTK * kTN + kTM * kTK;
    Element* sts_ptr = shm;

    // declare shared memory to register file copy plan.
    // tcu's wmma instruction prescribes a strict data to thread
    // mapping, in the current implementation, the shm-2-reg copy
    // plan is related to mma.
    typename KeTraits::TiledMma mma;
    typename KeTraits::TiledCopyG2S tiled_copy;

    auto rws = make_s2rA(sws_ptr, typename KeTraits::SmemLayoutA{}, mma);
    auto rxs = make_s2rB(sxs_ptr, typename KeTraits::SmemLayoutB{}, mma);
    auto rus = make_s2rA(sus_ptr, typename KeTraits::SmemLayoutC{}, mma);
    auto rhs = make_s2rB(shs_ptr, typename KeTraits::SmemLayoutD{}, mma);

    auto acc1 = get_acc<kTM, kTN>(mma);
    auto acc2 = get_acc<kTM, kTN>(mma);

    auto load_a_g2s_layout = tl::make_row_major_layout(kTM, kTK, kK);
    auto load_b_g2s_layout = tl::make_row_major_layout(kTN, kTK, kK);
    auto load_c_g2s_layout = tl::make_row_major_layout(kTM, kTK, kK);
    auto load_d_g2s_layout = tl::make_row_major_layout(kTN, kTK, kK);
    auto store_e_s2g_layout = tl::make_row_major_layout(kTM, kTN, kN);

    typename KeTraits::StoreE_R2S sts;  // declare register to shared store

    for (int k = 0; k < kK; k += kTK) {
        // TODO: Load data from global memory to shared memory
        copy_2d_tile_g2s(gws_ptr, sws_ptr, load_a_g2s_layout,
                         typename KeTraits::SmemLayoutA{}, tiled_copy);
        copy_2d_tile_g2s(gxs_ptr, sxs_ptr, load_b_g2s_layout,
                         typename KeTraits::SmemLayoutB{}, tiled_copy);
        copy_2d_tile_g2s(gus_ptr, sus_ptr, load_c_g2s_layout,
                         typename KeTraits::SmemLayoutC{}, tiled_copy);
        copy_2d_tile_g2s(ghs_ptr, shs_ptr, load_d_g2s_layout,
                         typename KeTraits::SmemLayoutD{}, tiled_copy);

        __copy_async();
        __syncthreads();

        for (int i = 0; i < rws.get_iters(); i++) {
            rws.copy(i);
            rxs.copy(i);
            gemm(mma, rws[i], rxs[i], acc1);
        }

        for (int i = 0; i < rus.get_iters(); i++) {
            rus.copy(i);
            rhs.copy(i);
            gemm(mma, rus[i], rhs[i], acc2);
        }

        __syncthreads();
        gws_ptr += kTK;
        gxs_ptr += kTK;
        gus_ptr += kTK;
        ghs_ptr += kTK;
    }

    __syncthreads();
    cute::axpby(1.0, acc1, 1.0, acc2);

    __syncthreads();
    if (current_block_x < total_block_x * 3 / 4) {
        cute_sigmoid(acc2);
    } else {
        cute_tanh(acc2);
    }
    __syncthreads();

    sts.copy(acc2, shm);

    __syncthreads();

    copy_2d_tile_s2g(sts_ptr, gts_ptr, typename KeTraits::SmemLayoutE{},
                     store_e_s2g_layout, typename KeTraits::TiledCopyS2G{});
}

template <typename Element>
__global__ void lstm_element_wise(const Element* i, const Element* f,
                                  const Element* o, const Element* c_candidate,
                                  const Element* c, Element* c_out,
                                  Element* h_out, int block_size, int size) {
    int index = blockIdx.x * block_size + threadIdx.x;

    if (index < size) {
        // TODO: Loading data into shared memory and computing, versus
        // computing directly in global memory, does not seem to make a
        // difference. This seems to require further optimization, such as
        // reconsidering redistributing data to different threads and performing
        // vectorized loading and storing.

        // This is a very naive kernel that loads data into shared memory and
        // then performs computations. It has been temporarily commented out.

        c_out[index] = f[index] * c[index] + i[index] * c_candidate[index];

        __syncthreads();

        h_out[index] = o[index] * tanh(c_out[index]);
    }
}

template <typename Element, typename CtaTileShape>
void lstm_gate(const Element* w, const Element* x, const Element* u,
               const Element* h, Element* t, const int m, const int n,
               const int k) {
    // Whole GEMM shape
    static const int kM = m;
    static const int kN = n;

    // CTA GEMM shape
    static const int kTM = dim_size<0, CtaTileShape>;
    static const int kTN = dim_size<1, CtaTileShape>;
    static const int kTK = dim_size<2, CtaTileShape>;

    using KeTraits = traits::DynLstmGateTraits<Element, CtaTileShape>;

    static constexpr int smem_size =
        std::max(kTK * (kTN + kTM) * 2, kTM * kTN) * sizeof(Element);

    auto lstm_gate = &dyn_lstm_gate<Element, KeTraits>;

    // maximal statically allocated smem per block
    const int kMaxSmemPerBlock = 48 * 1024;
    if (smem_size > kMaxSmemPerBlock) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(
            lstm_gate), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size);
    }

    const int block_m = (kM + kTM - 1) / kTM;
    const int block_n = (kN + kTN - 1) / kTN;

#ifdef DEBUG
    std::cout << "block_m: " << block_m << ", block_n: " << block_n
              << std::endl;
#endif

    const int kThreads = KeTraits::kThreads;

    dim3 gridDim(block_m, block_n, 1);
    dim3 blockDim(kThreads, 1, 1);

    lstm_gate<<<gridDim, blockDim, smem_size>>>(w, u, x, h, t, m, n, k);
}

template <typename Element, typename CtaTileShape>
void lstm_cell(const Element* w, const Element* x, const Element* u,
               const Element* c, const Element* h, Element* c_out,
               Element* h_out, int m, int n, int k) {
    static const int kM = m;
    static const int kN = n;

    static const int M = kM / 4;
    static const int N = kN;

    // Cuda malloc for output
    Element* t;
    CudaCheck(hipMalloc(&t, m * n * sizeof(Element)));

    lstm_gate<Element, CtaTileShape>(w, x, u, h, t, m, n, k);

    const Element* i = t;
    const Element* f = t + M * N;
    const Element* o = t + 2 * M * N;
    const Element* c_candidate = t + 3 * M * N;

    auto element_wise = &lstm_element_wise<Element>;

    /*
    TODO: Use `kMaxThreads` will case a runtime error:
    ```
    RuntimeError: CUDA error: invalid configuration argument
    CUDA kernel errors might be asynchronously reported at some other API call,
    so the stacktrace below might be incorrect. For debugging consider passing
    CUDA_LAUNCH_BLOCKING=1. Compile with `TORCH_USE_CUDA_DSA` to enable
    device-side assertions.
    ```
    */
    // int kMaxThreads = GetGPUMaxThreadsPerMultiProcessor(0);
    int size = M * N;
    int block_threads = 512;
    int block_size = (size + block_threads - 1) / block_threads;
    dim3 element_wise_grid_dim(block_size, 1, 1);
    dim3 element_wise_block_dim(block_threads, 1, 1);

    element_wise<<<element_wise_grid_dim, element_wise_block_dim>>>(
        i, f, o, c_candidate, c, c_out, h_out, block_threads, size);

    CudaCheck(hipFree(t));
}

void custom_lstm_cell_op(const torch::Tensor& w, const torch::Tensor& x,
                         const torch::Tensor& u, const torch::Tensor& c0,
                         const torch::Tensor& h0, torch::Tensor& c1,
                         torch::Tensor& h1, int64_t batch_size,
                         int64_t hidden_size) {
    using CtaTileShape = cell::TileShape<16, 32, 32>;

    auto dtype = w.dtype();

    int m = 4 * hidden_size;
    int n = batch_size;
    int k = hidden_size;

    if (dtype == torch::kHalf) {
        lstm_cell<cutlass::half_t, CtaTileShape>(
            reinterpret_cast<const cutlass::half_t*>(w.const_data_ptr()),
            reinterpret_cast<const cutlass::half_t*>(x.const_data_ptr()),
            reinterpret_cast<const cutlass::half_t*>(u.const_data_ptr()),
            reinterpret_cast<const cutlass::half_t*>(c0.const_data_ptr()),
            reinterpret_cast<const cutlass::half_t*>(h0.const_data_ptr()),
            reinterpret_cast<cutlass::half_t*>(c1.mutable_data_ptr()),
            reinterpret_cast<cutlass::half_t*>(h1.mutable_data_ptr()), m, n, k);
    } else {
        throw NotImplementedException("Unsupported data type.");
    }
}
}  // namespace tiledcuda::kernels
