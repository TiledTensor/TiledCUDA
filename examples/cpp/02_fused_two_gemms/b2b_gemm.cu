#include "hip/hip_runtime.h"
#include "b2b_gemm.hpp"
#include "util.hpp"

template <typename WholeShape, typename CtaTileShape, typename WarpLayout,
          const int kBatch>
void run(float epsilon = 1e-3) {
    using InType = __half;
    using AccType = float;

    static constexpr int kM = dim_size<0, WholeShape>;
    static constexpr int kN = dim_size<1, WholeShape>;
    static constexpr int kK = dim_size<2, WholeShape>;
    static constexpr int kP = dim_size<3, WholeShape>;

    static constexpr int kTM = dim_size<0, CtaTileShape>;
    static constexpr int kTN = dim_size<1, CtaTileShape>;
    static constexpr int kTK = dim_size<2, CtaTileShape>;
    static constexpr int kTP = dim_size<3, CtaTileShape>;

    static_assert(kK == kTK, "The current implementation requires kTK == K.");
    static_assert(kP == kTP, "The current implementation requires kTP == P.");

    thrust::host_vector<InType> h_a(kM * kK * kBatch);

    for (int i = 0; i < h_a.size(); ++i) {
        h_a[i] = static_cast<InType>(rand_float());
    }

    thrust::host_vector<InType> h_b(kK * kN * kBatch);
    for (int i = 0; i < h_b.size(); ++i) {
        h_b[i] = static_cast<InType>(rand_float());
    }

    thrust::host_vector<InType> h_c(kN * kP * kBatch);
    for (int i = 0; i < h_c.size(); ++i) {
        h_c[i] = static_cast<InType>(rand_float());
    }

    thrust::host_vector<AccType> h_d(kM * kP * kBatch);
    thrust::fill(h_d.begin(), h_d.end(), 0.);

    thrust::device_vector<InType> d_a = h_a;
    thrust::device_vector<InType> d_b = h_b;
    thrust::device_vector<InType> d_c = h_c;
    thrust::device_vector<AccType> d_d = h_d;

    const InType* A = thrust::raw_pointer_cast(d_a.data());
    const InType* B = thrust::raw_pointer_cast(d_b.data());
    const InType* C = thrust::raw_pointer_cast(d_c.data());
    AccType* D = thrust::raw_pointer_cast(d_d.data());

    using Config =
        B2BGemmTraits<InType, AccType, WholeShape, CtaTileShape, WarpLayout>;

    using RegA = typename Config::RegA;
    using RegB = typename Config::RegB;
    using RegC = typename Config::RegC;
    using RegD = typename Config::RegD;
    using RegAcc = typename Config::RegAcc;
    using RegAccCast = typename Config::RegAccCast;

    using GIteratorA = typename Config::GIteratorA;
    using SharedA = typename Config::SharedA;
    using SharedALoader = typename Config::SharedALoader;
    using RegALoader = typename Config::RegALoader;

    using GIteratorB = typename Config::GIteratorB;
    using SharedB = typename Config::SharedB;
    using SharedBLoader = typename Config::SharedBLoader;
    using RegBLoader = typename Config::RegBLoader;

    using GIteratorC = typename Config::GIteratorC;
    using SharedC = typename Config::SharedC;
    using SharedCLoader = typename Config::SharedCLoader;
    using RegCLoader = typename Config::RegCLoader;

    using DStorer = typename Config::DStorer;

    using ConvertAcc = typename Config::ConvertHalf;

    int block_x = CeilDiv<kM, kTM>;
    int block_y = CeilDiv<kP, kTP>;
    int block_z = kBatch;

    dim3 grid(block_x, block_y, block_z);
    dim3 block(Config::kThreads, 1, 1);

    int shm_input = (kTM * kTK + kTK * kTN + kTN * kTP);
    int shm_output = kTM * kTP;
    int shm_size = shm_input < shm_output ? shm_output * sizeof(InType)
                                          : shm_input * sizeof(InType);

    auto kernel = &KeBack2BackGemm<InType, AccType,            //
                                   GIteratorA, SharedA, RegA,  //
                                   SharedALoader, RegALoader,  //
                                   GIteratorB, SharedB, RegB,  //
                                   SharedBLoader, RegBLoader,  //
                                   GIteratorC, SharedC, RegC,  //
                                   SharedCLoader, RegCLoader,  //
                                   RegAcc, RegAccCast, typename Config::GlobalD,
                                   RegD, DStorer, ConvertAcc>;

    if (shm_size > 48 * 1024) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(
            kernel), hipFuncAttributeMaxDynamicSharedMemorySize, shm_size);
    }

    kernel<<<grid, block, shm_size, 0>>>(A, B, C, D, kM, kN, kK, kP, kTM, kTN,
                                         kTK, kTP);
    hipDeviceSynchronize();

    h_d = d_d;

    thrust::host_vector<InType> h_acc(kM * kN * kBatch);
    thrust::fill(h_acc.begin(), h_acc.end(), 0.);
    thrust::device_vector<InType> d_acc = h_acc;

    thrust::host_vector<InType> h_d2(kM * kP * kBatch);
    thrust::fill(h_d2.begin(), h_d2.end(), 0.);
    thrust::device_vector<InType> d_d2 = h_d2;

    cublas_two_gemms(kM, kN, kK, kP, kBatch, A, B, C,
                     thrust::raw_pointer_cast(d_d2.data()),
                     thrust::raw_pointer_cast(d_acc.data()));
    hipDeviceSynchronize();
    h_acc = d_acc;
    h_d2 = d_d2;

    float* data = thrust::raw_pointer_cast(h_d.data());
    __half* ground_truth = thrust::raw_pointer_cast(h_d2.data());

#ifdef DEBUG
    printf("ours:\n");
    for (int i = 0; i < h_d.size(); ++i) {
        printf("%.3f, ", data[i]);
        if (i && (i + 1) % 16 == 0) printf("\n");
    }
    printf("\nground_truth:\n");
    for (int i = 0; i < h_d.size(); ++i) {
        printf("%.3f, ", __half2float(ground_truth[i]));
        if (i && (i + 1) % 16 == 0) printf("\n");
    }
#endif

    if (check_results(data, ground_truth, kM * kP, epsilon)) {
        std::cout << "[" << kM << ", " << kN << ", " << kK << ", " << kP
                  << "], batch = " << kBatch << ", passed." << std::endl;
    } else {
        std::cout << "[" << kM << ", " << kN << ", " << kK << ", " << kP
                  << "], batch = " << kBatch << ", failed." << std::endl;
    }
}

int main() {
    using WarpLayout0 = tl::RowMajor<1, 1>;
    run<B2BGemmShape<16 /*M*/, 16 /*N*/, 16 /*K*/, 16 /*P*/>,
        B2BGemmShape<16 /*kTM*/, 16 /*kTN*/, 16 /*kTK*/, 16 /*kTP*/>,
        WarpLayout0, 1>();

    run<B2BGemmShape<16 /*M*/, 32 /*N*/, 16 /*K*/, 32 /*P*/>,
        B2BGemmShape<16 /*kTM*/, 16 /*kTN*/, 16 /*kTK*/, 32 /*kTP*/>,
        WarpLayout0, 1>();

    using WarpLayout1 = tl::RowMajor<2, 1>;
    run<B2BGemmShape<32 /*M*/, 64 /*N*/, 32 /*K*/, 64 /*P*/>,
        B2BGemmShape<32 /*kTM*/, 32 /*kTN*/, 32 /*kTK*/, 64 /*kTP*/>,
        WarpLayout1, 1>();

    run<B2BGemmShape<64 /*M*/, 64 /*N*/, 32 /*K*/, 64 /*P*/>,
        B2BGemmShape<32 /*kTM*/, 32 /*kTN*/, 32 /*kTK*/, 64 /*kTP*/>,
        WarpLayout1, 1>();

    using WarpLayout2 = tl::RowMajor<4, 1>;
    run<B2BGemmShape<256 /*M*/, 128 /*N*/, 64 /*K*/, 64 /*P*/>,
        B2BGemmShape<64 /*kTM*/, 32 /*kTN*/, 64 /*kTK*/, 64 /*kTP*/>,
        WarpLayout1, 1>(5e-3);

    run<B2BGemmShape<1024 /*M*/, 1024 /*N*/, 128 /*K*/, 128 /*P*/>,
        B2BGemmShape<64 /*kTM*/, 64 /*kTN*/, 128 /*kTK*/, 128 /*kTP*/>,
        WarpLayout2, 1>(8e-2 /*epsilon*/);

    // batched
    run<B2BGemmShape<16 /*M*/, 16 /*N*/, 16 /*K*/, 16 /*P*/>,
        B2BGemmShape<16 /*kTM*/, 16 /*kTN*/, 16 /*kTK*/, 16 /*kTP*/>,
        WarpLayout0, 2>();

    run<B2BGemmShape<1024 /*M*/, 1024 /*N*/, 128 /*K*/, 128 /*P*/>,
        B2BGemmShape<64 /*kTM*/, 64 /*kTN*/, 128 /*kTK*/, 128 /*kTP*/>,
        WarpLayout2, 5>(8e-2 /*epsilon*/);

    run<B2BGemmShape<64 /*M*/, 256 /*N*/, 128 /*K*/, 128 /*P*/>,
        B2BGemmShape<64 /*kTM*/, 64 /*kTN*/, 128 /*kTK*/, 128 /*kTP*/>,
        WarpLayout2, 1>(8e-2 /*epsilon*/);

    return 0;
}
