#include "hip/hip_runtime.h"
#include "fused_gemm.hpp"
#include "util.hpp"

#include <fstream>
#include <iomanip>
#include <sstream>
#include <string>

template <typename WholeShape, typename CtaTileShape, typename WarpLayout,
          const int kBatch>
void run(std::ofstream& fout, float epsilon = 1e-3, int iters = 20) {
    using InType = __half;
    using AccType = float;

    static constexpr int kM = dim_size<0, WholeShape>;
    static constexpr int kN = dim_size<1, WholeShape>;
    static constexpr int kK = dim_size<2, WholeShape>;
    static constexpr int kP = dim_size<3, WholeShape>;

    static constexpr int kTM = dim_size<0, CtaTileShape>;
    static constexpr int kTN = dim_size<1, CtaTileShape>;
    static constexpr int kTK = dim_size<2, CtaTileShape>;
    static constexpr int kTP = dim_size<3, CtaTileShape>;

    static_assert(kK == kTK, "The current implementation requires kTK == K.");
    static_assert(kP == kTP, "The current implementation requires kTP == P.");

    thrust::host_vector<InType> h_a(kM * kK * kBatch);

    for (int i = 0; i < h_a.size(); ++i) {
        h_a[i] = static_cast<InType>(rand_float());
    }

    thrust::host_vector<InType> h_b(kK * kN * kBatch);
    for (int i = 0; i < h_b.size(); ++i) {
        h_b[i] = static_cast<InType>(rand_float());
    }

    thrust::host_vector<InType> h_c(kN * kP * kBatch);
    for (int i = 0; i < h_c.size(); ++i) {
        h_c[i] = static_cast<InType>(rand_float());
    }

    thrust::host_vector<AccType> h_d(kM * kP * kBatch);
    thrust::fill(h_d.begin(), h_d.end(), 0.);

    thrust::device_vector<InType> d_a = h_a;
    thrust::device_vector<InType> d_b = h_b;
    thrust::device_vector<InType> d_c = h_c;
    thrust::device_vector<AccType> d_d = h_d;

    const InType* A = thrust::raw_pointer_cast(d_a.data());
    const InType* B = thrust::raw_pointer_cast(d_b.data());
    const InType* C = thrust::raw_pointer_cast(d_c.data());
    AccType* D = thrust::raw_pointer_cast(d_d.data());

    using Config =
        FusedGemmTraits<InType, AccType, WholeShape, CtaTileShape, WarpLayout>;

    using RegA = typename Config::RegA;
    using RegB = typename Config::RegB;
    using RegC = typename Config::RegC;
    using RegD = typename Config::RegD;
    using RegAcc = typename Config::RegAcc;
    using RegAccCast = typename Config::RegAccCast;

    using GIteratorA = typename Config::GIteratorA;
    using SharedA = typename Config::SharedA;
    using SharedALoader = typename Config::SharedALoader;
    using RegALoader = typename Config::RegALoader;

    using GIteratorB = typename Config::GIteratorB;
    using SharedB = typename Config::SharedB;
    using SharedBLoader = typename Config::SharedBLoader;
    using RegBLoader = typename Config::RegBLoader;

    using GIteratorC = typename Config::GIteratorC;
    using SharedC = typename Config::SharedC;
    using SharedCLoader = typename Config::SharedCLoader;
    using RegCLoader = typename Config::RegCLoader;

    using DStorer = typename Config::DStorer;

    using ConvertAcc = typename Config::ConvertHalf;

    int block_x = CeilDiv<kM, kTM>;
    int block_y = CeilDiv<kP, kTP>;
    int block_z = kBatch;

    dim3 grid(block_x, block_y, block_z);
    dim3 block(Config::kThreads, 1, 1);

    int shm_input = (kTM * kTK + kTK * kTN + kTN * kTP);
    int shm_output = kTM * kTP;
    int shm_size = shm_input < shm_output ? shm_output * sizeof(InType)
                                          : shm_input * sizeof(InType);

    auto kernel = &KeFusedGemm<InType, AccType,            //
                               GIteratorA, SharedA, RegA,  //
                               SharedALoader, RegALoader,  //
                               GIteratorB, SharedB, RegB,  //
                               SharedBLoader, RegBLoader,  //
                               GIteratorC, SharedC, RegC,  //
                               SharedCLoader, RegCLoader,  //
                               RegAcc, RegAccCast, typename Config::GlobalD,
                               RegD, DStorer, ConvertAcc>;

    if (shm_size > 48 * 1024) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(
            kernel), hipFuncAttributeMaxDynamicSharedMemorySize, shm_size);
    }

    kernel<<<grid, block, shm_size, 0>>>(A, B, C, D, kM, kN, kK, kP, kTM, kTN,
                                         kTK, kTP);
    hipDeviceSynchronize();

    CudaCheckLastError();

    h_d = d_d;

    thrust::host_vector<InType> h_acc(kM * kN * kBatch);
    thrust::fill(h_acc.begin(), h_acc.end(), 0.);
    thrust::device_vector<InType> d_acc = h_acc;

    thrust::host_vector<InType> h_d2(kM * kP * kBatch);
    thrust::fill(h_d2.begin(), h_d2.end(), 0.);
    thrust::device_vector<InType> d_d2 = h_d2;

    cublas_two_gemms(kM, kN, kK, kP, kBatch, A, B, C,
                     thrust::raw_pointer_cast(d_d2.data()),
                     thrust::raw_pointer_cast(d_acc.data()), false);
    hipDeviceSynchronize();

    CudaCheckLastError();

    h_acc = d_acc;
    h_d2 = d_d2;

    float* data = thrust::raw_pointer_cast(h_d.data());
    __half* ground_truth = thrust::raw_pointer_cast(h_d2.data());

#ifdef DEBUG
    printf("ours:\n");
    for (int i = 0; i < h_d.size(); ++i) {
        printf("%.3f, ", data[i]);
        if (i && (i + 1) % 16 == 0) printf("\n");
    }
    printf("\nground_truth:\n");
    for (int i = 0; i < h_d.size(); ++i) {
        printf("%.3f, ", __half2float(ground_truth[i]));
        if (i && (i + 1) % 16 == 0) printf("\n");
    }
#endif

    bool passed = check_results(data, ground_truth, kM * kP * kBatch, epsilon);

    if (passed) {
        std::cout << "[" << kM << ", " << kN << ", " << kK << ", " << kP
                  << "], batch = " << kBatch << ", passed." << std::endl;
    } else {
        std::cout << "[" << kM << ", " << kN << ", " << kK << ", " << kP
                  << "], batch = " << kBatch << ", failed." << std::endl;
    }

    if (passed) {
        int warm_up = 10;
        for (int i = 0; i < warm_up; ++i)
            kernel<<<grid, block, shm_size, 0>>>(A, B, C, D, kM, kN, kK, kP,
                                                 kTM, kTN, kTK, kTP);
        hipDeviceSynchronize();

        CudaTimer timer;
        timer.start();
        for (int i = 0; i < iters; ++i) {
            kernel<<<grid, block, shm_size, 0>>>(A, B, C, D, kM, kN, kK, kP,
                                                 kTM, kTN, kTK, kTP);
        }
        hipDeviceSynchronize();
        float time2 = timer.stop() / iters;

        float time1 =
            cublas_two_gemms(kM, kN, kK, kP, kBatch, A, B, C,
                             thrust::raw_pointer_cast(d_d2.data()),
                             thrust::raw_pointer_cast(d_acc.data()), true);

        std::cout << "cuBLAS\tTiledCUDA\tRatio" << std::endl;
        std::cout << std::setprecision(4) << time1 << "\t" << time2 << "\t"
                  << time2 / time1 << std::endl;

        fout << "[" << kM << ", " << kN << ", " << kK << ", " << kP << ", "
             << kBatch << "], " << "\t";
        fout << "[" << kTM << ", " << kTN << ", " << kTK << ", " << kTP << "], "
             << "\t";
        fout << time1 << "\t" << time2 << "\t" << time2 / time1 << std::endl;
    } else {
        std::cerr << "Test failed." << std::endl;
    }
}

int main() {
    std::ofstream fout;
    fout.setf(std::ios::fixed);
    fout.precision(4);

    std::stringstream file_name;
    file_name << "fused_gemm.tsv";
    fout.open(file_name.str(), std::ios::out);

    fout << "[M, N, K, P, kBatch]" << "\t" << "[kTM, kTN, kTK, kTP]" << "\t"
         << "cuBLAS" << "\t" << "TiledCUDA" << "\t" << "Ratio" << std::endl;

    using WarpLayout0 = tl::RowMajor<1, 1>;
    run<FusedGemmShape<16 /*M*/, 16 /*N*/, 16 /*K*/, 16 /*P*/>,
        FusedGemmShape<16 /*kTM*/, 16 /*kTN*/, 16 /*kTK*/, 16 /*kTP*/>,
        WarpLayout0, 1>(fout);

    run<FusedGemmShape<16 /*M*/, 32 /*N*/, 16 /*K*/, 32 /*P*/>,
        FusedGemmShape<16 /*kTM*/, 16 /*kTN*/, 16 /*kTK*/, 32 /*kTP*/>,
        WarpLayout0, 1>(fout);

    using WarpLayout1 = tl::RowMajor<2, 1>;
    run<FusedGemmShape<32 /*M*/, 64 /*N*/, 32 /*K*/, 64 /*P*/>,
        FusedGemmShape<32 /*kTM*/, 32 /*kTN*/, 32 /*kTK*/, 64 /*kTP*/>,
        WarpLayout1, 1>(fout);

    run<FusedGemmShape<64 /*M*/, 64 /*N*/, 32 /*K*/, 64 /*P*/>,
        FusedGemmShape<32 /*kTM*/, 32 /*kTN*/, 32 /*kTK*/, 64 /*kTP*/>,
        WarpLayout1, 1>(fout);

    using WarpLayout2 = tl::RowMajor<4, 1>;

    run<FusedGemmShape<256 /*M*/, 128 /*N*/, 64 /*K*/, 64 /*P*/>,
        FusedGemmShape<64 /*kTM*/, 32 /*kTN*/, 64 /*kTK*/, 64 /*kTP*/>,
        WarpLayout1, 1>(fout, 5e-3);

    run<FusedGemmShape<1024 /*M*/, 1024 /*N*/, 128 /*K*/, 128 /*P*/>,
        FusedGemmShape<64 /*kTM*/, 64 /*kTN*/, 128 /*kTK*/, 128 /*kTP*/>,
        WarpLayout2, 1>(fout, 8e-2 /*epsilon*/);

    // batched
    run<FusedGemmShape<16 /*M*/, 16 /*N*/, 16 /*K*/, 16 /*P*/>,
        FusedGemmShape<16 /*kTM*/, 16 /*kTN*/, 16 /*kTK*/, 16 /*kTP*/>,
        WarpLayout0, 2>(fout);

    run<FusedGemmShape<1024 /*M*/, 1024 /*N*/, 128 /*K*/, 128 /*P*/>,
        FusedGemmShape<64 /*kTM*/, 64 /*kTN*/, 128 /*kTK*/, 128 /*kTP*/>,
        WarpLayout2, 5>(fout, 8e-2 /*epsilon*/);

    run<FusedGemmShape<64 /*M*/, 256 /*N*/, 128 /*K*/, 128 /*P*/>,
        FusedGemmShape<64 /*kTM*/, 64 /*kTN*/, 128 /*kTK*/, 128 /*kTP*/>,
        WarpLayout2, 1>(fout, 8e-2 /*epsilon*/);

    run<FusedGemmShape<2048 /*M*/, 1024 /*N*/, 128 /*K*/, 128 /*P*/>,
        FusedGemmShape<64 /*kTM*/, 128 /*kTN*/, 128 /*kTK*/, 128 /*kTP*/>,
        WarpLayout2, 1>(fout, 8e-2 /*epsilon*/);

    run<FusedGemmShape<1024 /*M*/, 2048 /*N*/, 128 /*K*/, 128 /*P*/>,
        FusedGemmShape<64 /*kTM*/, 128 /*kTN*/, 128 /*kTK*/, 128 /*kTP*/>,
        WarpLayout2, 1>(fout, 8e-2 /*epsilon*/);

    // Test failed.
    // run<FusedGemmShape<2048 /*M*/, 2048 /*N*/, 128 /*K*/, 128 /*P*/>,
    //     FusedGemmShape<64 /*kTM*/, 128 /*kTN*/, 128 /*kTK*/, 128 /*kTP*/>,
    //     WarpLayout2, 1>(fout, 8e-2 /*epsilon*/);

    // CUDA error: misaligned address in tiledcuda fused_gemm.
    // run<FusedGemmShape<2048 /*M*/, 2048 /*N*/, 128 /*K*/, 128 /*P*/>,
    //     FusedGemmShape<128 /*kTM*/, 128 /*kTN*/, 128 /*kTK*/, 128 /*kTP*/>,
    //     WarpLayout2, 1>(fout, 8e-2 /*epsilon*/);

    return 0;
}
