#include "hip/hip_runtime.h"
#include "gemm.hpp"
#include "util.hpp"

template <typename WholeShape, typename CtaTileShape, typename WarpLayout>
int run_test() {
    using InType = __half;
    using AccType = float;

    static constexpr int kM = dim_size<0, WholeShape>;
    static constexpr int kN = dim_size<1, WholeShape>;
    static constexpr int kK = dim_size<2, WholeShape>;

    static constexpr int kTM = dim_size<0, CtaTileShape>;
    static constexpr int kTN = dim_size<1, CtaTileShape>;

    thrust::host_vector<InType> h_a(kM * kK);
    for (int i = 0; i < h_a.size(); ++i)
        h_a[i] = static_cast<InType>(rand_float());

    thrust::host_vector<InType> h_b(kK * kN);
    for (int i = 0; i < h_b.size(); ++i)
        h_b[i] = static_cast<InType>(rand_float());

    thrust::host_vector<AccType> h_c(kM * kN);
    thrust::fill(h_c.begin(), h_c.end(), 0.);

    thrust::device_vector<InType> d_a = h_a;
    thrust::device_vector<InType> d_b = h_b;
    thrust::device_vector<AccType> d_c = h_c;

    const InType* A = thrust::raw_pointer_cast(d_a.data());
    const InType* B = thrust::raw_pointer_cast(d_b.data());
    AccType* C = thrust::raw_pointer_cast(d_c.data());

    using Config =
        GemmTraits<InType, AccType, WholeShape, kTM, kTN, WarpLayout>;

    using RegA = typename Config::RegA;
    using RegB = typename Config::RegB;
    using RegC = typename Config::RegC;

    using IteratorA = typename Config::IteratorA;
    using IteratorB = typename Config::IteratorB;

    int block_x = CeilDiv<kM, kTM>;
    int block_y = CeilDiv<kN, kTN>;

    std::cout << "kThreads: " << Config::kThreads << std::endl
              << "RegA: " << RegA{} << std::endl
              << "RegB: " << RegB{} << std::endl
              << "RegC: " << RegC{} << std::endl
              << "IteratorA: " << IteratorA{} << std::endl
              << "IteratorB: " << IteratorB{} << std::endl
              << "blocks: [" << block_x << ", " << block_y << "]" << std::endl
              << std::endl;

    dim3 dim_grid(block_x, block_y, 1);
    dim3 dim_block(Config::kThreads, 1, 1);
    simple_gemm<InType, AccType, kM, kN, kK, kTM, kTN, IteratorA, RegA,
                typename Config::ALoader, IteratorB, RegB,
                typename Config::BLoader, typename Config::GlobalC,
                typename Config::RegC, typename Config::CStorer>
        <<<dim_grid, dim_block>>>(A, B, C);
    hipDeviceSynchronize();
    h_c = d_c;

    // check correctness
    thrust::device_vector<InType> d_c2(kM * kN);
    thrust::fill(d_c2.begin(), d_c2.end(), 0.);

    cublas_hgemm(kM, kN, kK, thrust::raw_pointer_cast(d_a.data()),
                 thrust::raw_pointer_cast(d_b.data()),
                 thrust::raw_pointer_cast(d_c2.data()), false /*timeit*/);
    thrust::host_vector<InType> h_c2 = d_c2;

    bool passed = check_results(thrust::raw_pointer_cast(h_c.data()),
                                thrust::raw_pointer_cast(h_c2.data()), kM * kN);

    if (passed) {
        std::cout << "Test passed." << std::endl;

        CudaTimer timer;
        timer.start();
        int iters = 20;
        for (int i = 0; i < iters; ++i) {
            simple_gemm<InType, AccType, kM, kN, kK, kTM, kTN, IteratorA, RegA,
                        typename Config::ALoader, IteratorB, RegB,
                        typename Config::BLoader, typename Config::GlobalC,
                        typename Config::RegC, typename Config::CStorer>
                <<<dim_grid, dim_block>>>(A, B, C);
        }
        hipDeviceSynchronize();

        float time = timer.stop();
        std::cout << std::setprecision(4) << "elapsed time: " << time / iters
                  << " ms" << std::endl;

    } else
        std::cerr << "Test failed." << std::endl;

    return 0;
}

int main(int argc, char* argv[]) {
    run_test<GemmShape<1024, 1024, 2048>, GemmShape<256, 128, 64>,
             tl::RowMajor<2, 2>>();

    return 0;
}
