#include "hip/hip_runtime.h"
#include "cell/mod.hpp"
#include "common/test_utils.hpp"
#include "types/tile_iterator.hpp"
#include "types/types.hpp"

#include <glog/logging.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

namespace tiledcuda {

using namespace cell;
using namespace cute;

namespace tl = tile_layout;

namespace {

template <typename DType1, typename DType2>
void check_result1(const DType1* a, const DType1* b, DType2* c, int M, int N,
                   int K, const float* result) {
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            DType2 s = 0.;
            for (int k = 0; k < K; ++k) s += a[i * K + k] * b[k * N + j];
            c[i * N + j] = s;
        }
    }

    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            EXPECT_NEAR(c[i * N + j], result[i * N + j], 1e-2);
        }
    }
}

/// utility function
template <typename Element, const int kRows, const int kCols>
__device__ void print_tile(const Element* data, int delimeter = kCols) {
    if (threadIdx.x || blockIdx.x || blockIdx.y) return;

    for (int i = 0; i < kRows * kCols; ++i) {
        printf("%.2f, ", float(data[i] * 1.0_hf));  // print cutlass::half_t
        if ((i + 1) % delimeter == 0) printf("\n");
    }
    printf("\n");
}

template <typename Element, typename ElementAcc, typename LoadSharedA,
          typename LoadSharedB, typename StoreSharedC, typename TileIteratorA,
          typename TileIteratorB, typename SharedC, typename WarpLayout>
__global__ void test_wmma1(const Element* ga, const Element* gb,
                           ElementAcc* gc) {
    extern __shared__ __align__(sizeof(double)) unsigned char buf_[];
    auto* shared_a = reinterpret_cast<Element*>(buf_);
    auto* shared_b = shared_a + TileIteratorA::Tile::kNumel;
    auto* shared_c = reinterpret_cast<ElementAcc*>(buf_);

    // transfer data tiles from global to shared
    copy::copy_2d_tile_g2s(ga, shared_a, typename LoadSharedA::SrcLayout{},
                           typename LoadSharedA::DstLayout{},
                           typename LoadSharedA::TiledCopy{});
    copy::copy_2d_tile_g2s(gb, shared_b, typename LoadSharedB::SrcLayout{},
                           typename LoadSharedB::DstLayout{},
                           typename LoadSharedB::TiledCopy{});
    __copy_async();
    __syncthreads();

    Reg<Element, tl::RowMajor<4, 24>> rA;
    Reg<Element, tl::RowMajor<4, 24>> rB;
    Reg<ElementAcc, tl::RowMajor<2, 8>> acc;

    TileIteratorA sAs(shared_a);
    TileIteratorB sBs(shared_b);

    static_assert(TileIteratorA::sc1 == TileIteratorB::sc0,
                  "dimension mismatch!");

    for (int k = 0; k < TileIteratorA::sc1; ++k) {
        copy::copy_tile_s2r(*sAs(_, k), rA, WarpLayout{});
        copy::copy_tile_s2r(*sBs(k, _), rB, WarpLayout{});

        compute::gemm_(rA, rB, acc);
    }
    __syncthreads();

    SharedC sC(shared_c);
    copy::copy_tile_r2s(acc, sC, WarpLayout{});
    __syncthreads();

    copy::copy_2d_tile_s2g(shared_c, gc, typename StoreSharedC::SrcLayout{},
                           typename StoreSharedC::DstLayout{},
                           typename StoreSharedC::TiledCopy{});
}

}  // namespace

namespace testing {

TEST(TestWmma, shape1) {
    // unittest for register-level gemm by calling into wmma PTX
    using Element = cutlass::half_t;
    using ElementAcc = float;

    using WarpLayout = tl::RowMajor<2, 2>;
    static const int kThreads = tl::get_numel<WarpLayout> * 32;

    LOG(INFO) << "kThreads: " << kThreads << std::endl;

    // M, N, K for shared memory tile
    const int M = 64;
    const int N = 64;
    const int K = 128;

    thrust::host_vector<Element> h_a(M * K);
    thrust::host_vector<Element> h_b(K * N);
    thrust::host_vector<ElementAcc> h_c(M * N);

    for (int i = 0; i < h_a.size(); ++i)
        h_a[i] = static_cast<Element>(i % 2048 / 100.);

    for (int i = 0; i < h_b.size(); ++i)
        h_b[i] = static_cast<Element>(i % 2048 / 100.);

    thrust::fill(h_c.begin(), h_c.end(), 0.);

    thrust::device_vector<Element> d_a = h_a;
    thrust::device_vector<Element> d_b = h_b;
    thrust::device_vector<ElementAcc> d_c = h_c;

    dim3 dim_grid(1, 1, 1);
    dim3 dim_block(kThreads, 1, 1);

    int size_ab = (M + N) * K * sizeof(Element);
    int size_c = M * N * sizeof(ElementAcc);
    int shm_size = size_ab > size_c ? size_ab : size_c;

    using SharedA = Shared<Element, tl::RowMajor<M, K>>;
    using TileIteratorA = TileIterator<SharedA, TileShape<32, 32>>;
    LOG(INFO) << "TileIteratorA: [" << TileIteratorA::Tile::kRows << ", "
              << TileIteratorA::Tile::kCols
              << "]; numel = " << TileIteratorA::Tile::kNumel << std::endl
              << "sc0 = " << TileIteratorA::sc0
              << ", sc1 = " << TileIteratorA::sc1 << std::endl;

    using SharedB = Shared<Element, tl::RowMajor<K, N>>;
    using TileIteratorB = TileIterator<SharedB, TileShape<32, 32>>;
    LOG(INFO) << "TileIteratorB: sc0 = " << TileIteratorB::sc0
              << ", sc1 = " << TileIteratorB::sc1 << std::endl;

    using SharedC = Shared<ElementAcc, tl::RowMajor<M, N>>;

    // for global to shared memory copy using CuTe
    using LoadSharedA = traits::G2S2DCopyTraits<Element, M, K, M, K, kThreads,
                                                false /*use swizzle*/>;
    using LoadSharedB = traits::G2S2DCopyTraits<Element, N, K, N, K, kThreads,
                                                false /*use swizzle*/>;
    // transfer operand C from shared memory to global memory
    using StoreSharedC =
        traits::S2G2DCopyTraits<ElementAcc, M, N, M, N, kThreads,
                                false /*use swizzle*/>;

    test_wmma1<Element, ElementAcc, LoadSharedA, LoadSharedB, StoreSharedC,
               TileIteratorA, TileIteratorB, SharedC, WarpLayout>
        <<<dim_grid, dim_block, shm_size>>>(
            thrust::raw_pointer_cast(d_a.data()),
            thrust::raw_pointer_cast(d_b.data()),
            thrust::raw_pointer_cast(d_c.data()));

    hipDeviceSynchronize();

    thrust::host_vector<float> h_c1;
    h_c1 = d_c;

    // {  // ground truth

    //     check_result1(thrust::raw_pointer_cast(h_a.data()),
    //                   thrust::raw_pointer_cast(h_b.data()),
    //                   thrust::raw_pointer_cast(h_c.data()), M, N, K,
    //                   thrust::raw_pointer_cast(h_c1.data()));
    // }
}

}  // namespace testing
}  // namespace tiledcuda
