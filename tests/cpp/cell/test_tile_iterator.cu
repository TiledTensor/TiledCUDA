#include "hip/hip_runtime.h"
#include "common/test_utils.hpp"
#include "types/mod.hpp"

namespace tiledcuda::testing {

using namespace cell;
namespace tl = tile_layout;

namespace {
template <typename Element, typename Layout>
__device__ void init_value(Element* data, Layout layout) {
    int count = 0;
    for (int i = 0; i < Layout::kRows; ++i) {
        for (int j = 0; j < Layout::kCols; ++j) {
            data[layout(i, j)] = static_cast<Element>(++count);
        }
    }
}

template <typename Iterator>
__global__ void test_tile_iterator() {
    using DType = typename Iterator::Tile::DType;

    extern __shared__ __align__(sizeof(double)) unsigned char buf_[];
    auto* buf = reinterpret_cast<DType*>(buf_);
    init_value(buf, typename Iterator::Tile::Layout{});

    typename Iterator::Tile s_tile(buf);
    printf("Shared tile:\n");
    s_tile.dump_value();

    Iterator tiles(buf);

    printf("\nIterate over rows.\n");
    for (int i = 0; i < Iterator::sc0; ++i) {
        printf("Iteration-[%d, _]:\n", i);
        tiles(i, _).to_tile().dump_value();
        printf("\n");
    }

    printf("Iterate over columns.\n\n");
    for (int j = 0; j < Iterator::sc1; ++j) {
        printf("Iteration-[_, %d]:\n", j);
        tiles(_, j).to_tile().dump_value();
        printf("\n");
    }

    printf("Iterate over rows and columns.\n\n");
    for (int i = 0; i < Iterator::sc0; ++i) {
        for (int j = 0; j < Iterator::sc1; ++j) {
            printf("Iteration-[%d, %d]:\n", i, j);
            tiles(i, j).dump_value();
            printf("\n");
        }
    }

    printf("Another way to iterate over rows and columns.\n\n");
    for (int i = 0; i < Iterator::sc0; ++i) {
        auto cols = tiles(i, _);

        printf("\n");
        for (int j = 0; j < decltype(cols)::sc1; ++j) {
            printf("Iteration-[%d, %d]:\n", i, j);
            cols(j).dump_value();
        }
    }

    printf("Another way to iterate over rows and columns.\n\n");
    for (int i = 0; i < Iterator::sc1; ++i) {
        auto rows = tiles(_, i);

        for (int j = 0; j < decltype(rows)::sc0; ++j) {
            printf("Iteration-[%d, %d]:\n", i, j);
            rows(j).dump_value();
        }
    }
}
}  // namespace

// FIXME(haruhi): Currently, these unit tests only output the values. Implement
// stricter and more meaningful correctness checks.
TEST(TestTile, test_row_major) {
    using Element = cutlass::half_t;

    const int rows = 4;
    const int cols = 12;

    using Tile = SharedTile<Element, tl::RowMajor<rows, cols>>;
    using Iterator = TileIterator<Tile, TileShape<2, 4>>;

    LOG(INFO) << std::endl << "Test Row-major" << std::endl;

    int shm_size = Tile::kNumel * sizeof(Element);
    // DONOT change this launch config. The unittest is implemented for a
    // single thread.
    test_tile_iterator<Iterator><<<1, 1, shm_size>>>();
    hipDeviceSynchronize();
}

TEST(TestTile, test_col_major) {
    using Element = cutlass::half_t;

    const int rows = 4;
    const int cols = 12;

    using Tile = SharedTile<Element, tl::ColMajor<rows, cols>>;
    using Iterator = TileIterator<Tile, TileShape<2, 4>>;

    LOG(INFO) << std::endl << "Test Column-major" << std::endl;

    int shm_size = Tile::kNumel * sizeof(Element);
    // DONOT change this launch config. The unittest is implemented for a
    // single thread.
    test_tile_iterator<Iterator><<<1, 1, shm_size>>>();
    hipDeviceSynchronize();
}

TEST(TestTile, test_swizzled_row_major) {
    using Element = float;

    const int rows = 16;
    const int cols = 16;

    const int chunked_row = 16;
    const int chunked_col = 16;

    using Layout = tl::RowMajor<rows, cols>;
    using Tile = SharedTile<Element, Layout, true>;
    using Iterator = TileIterator<Tile, TileShape<chunked_row, chunked_col>>;

    LOG(INFO) << std::endl << "Test Row-major" << std::endl;

    int shm_size = Tile::kNumel * sizeof(Element);
    // DONOT change this launch config. The unittest is implemented for a
    // single thread.
    test_tile_iterator<Iterator><<<1, 1, shm_size>>>();
    hipDeviceSynchronize();
}

}  // namespace tiledcuda::testing
