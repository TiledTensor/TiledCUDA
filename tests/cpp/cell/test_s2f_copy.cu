#include "hip/hip_runtime.h"
#include "cell/copy/mod.hpp"
#include "cell/sync.hpp"
#include "cell/traits/copy.hpp"
#include "common/test_utils.hpp"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

namespace traits = tiledcuda::cell::traits;

namespace tiledcuda {

namespace {
template <typename Element>
__device__ void init_data(Element* data, int64_t numel) {
    if (threadIdx.x == 0) {
        for (int i = 0; i < numel; ++i) {
            data[i] = static_cast<Element>(i);
        }
    }
}

template <typename Element>
__device__ void debug_print(Element* data, int rows, int cols) {
    if (threadIdx.x == 0) {
        printf("\ntile shape = [%d, %d]\n", rows, cols);
        for (int i = 0; i < rows; ++i) {
            printf("%d: ", i);

            for (int j = 0; j < cols - 1; ++j) {
                printf("%.0f, ", static_cast<float>(data[i * cols + j]));
            }
            printf("%.0f\n", static_cast<float>(data[(i + 1) * cols - 1]));
        }
        printf("\n");
    }
}

template <typename Element, typename S2RTraits>
__global__ void copy_s2r() {
    extern __shared__ __align__(sizeof(double)) unsigned char buf_[];
    auto* buf = reinterpret_cast<Element*>(buf_);

    typename S2RTraits::SrcLayout s_layout;
    typename S2RTraits::DstLayout d_layout;
    typename S2RTraits::ThreadLayout t_layout;

    Element* dst = nullptr;

    int rows = size<0>(s_layout);
    int cols = size<1>(s_layout);
    int numel = rows * cols;

    int tid = threadIdx.x;

    init_data(buf, numel);
    debug_print(buf, rows, cols);

    int64_t stride = 16;
    for (int k = 0; k < 2; ++k) {
        cell::copy::copy_2d_tile_s2r<Element, decltype(s_layout),
                                     decltype(d_layout), decltype(t_layout)>(
            buf + (k * stride) /*src_ptr*/, dst /*dst_ptr*/, tid);
    }
}
}  // namespace

namespace testing {
TEST(TestShm2Rf, copy_2d_tile_s2r) {
    using Element = cutlass::half_t;

    const int kRows = 16;
    const int kCols = 32;

    // swizzled row major layout for shared memory
    using Swizzled = SwizzledRowMajor<Element, kRows, kCols, 0>;
    using SrcLayout = typename Swizzled::SmemLayout;

    using DstLayout = RowMajor<kRows, kCols>;
    using ThreadLayout = RowMajor<16, 2>;

    dim3 dim_grid(1, 1, 1);
    dim3 dim_block(32, 1, 1);

    int shm_size = kRows * kCols * sizeof(Element);

    using S2RCopyTraits =
        traits::S2R2DCopyTraits<Element, SrcLayout, DstLayout, ThreadLayout>;
    copy_s2r<Element, S2RCopyTraits><<<dim_grid, dim_block, shm_size>>>();
    hipDeviceSynchronize();
}

}  // namespace testing
}  // namespace tiledcuda
