#include "hip/hip_runtime.h"
#include "cell/copy/mod.hpp"
#include "cell/sync.hpp"
#include "common/test_utils.hpp"
#include "types/mod.hpp"

#include <glog/logging.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <sstream>

namespace tiledcuda::testing {
using namespace cell;
using namespace copy;
namespace tl = tile_layout;

namespace {
template <typename Element>
__device__ void init_value(Element* data, int numel) {
    for (int i = 0; i < numel; ++i) {
        data[i] = static_cast<Element>(0.);
    }
}

__global__ void init_halfs(__half* data, int64_t numel) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < numel) {
        data[tid] = __float2half(tid % 2048);
    }
}

template <typename Reg, typename DType>
DEVICE void check_results(const Reg& r_tile, const Reg& r_tile_swizzled,
                          int rows, int cols) {
    const int numel = BaseTileRowMajor<DType>::kNumel;

    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            const DType* data1 = r_tile(i, j).data();
            const DType* data2 = r_tile_swizzled(i, j).data();

            for (int n = 0; n < numel; ++n) assert(data1[n] == data2[n]);
        }
    }
}

template <typename Element, typename Global, typename GIterator,
          typename Shared1, typename SIterator1, typename Shared2,
          typename SIterator2, typename Reg, typename G2S1, typename G2S2,
          typename S2R>
__global__ void swizzled_copy(const Element* data, G2S1& g2s,
                              G2S2& g2s_swizzled, S2R& s2r) {
    extern __shared__ __align__(sizeof(double)) unsigned char buf_[];
    auto* buf = reinterpret_cast<Element*>(buf_);
    init_value(buf, Shared1::kNumel + Shared2::kNumel);

    GIterator g_tiles(data);

    Shared1 s_tile(buf);
    Shared2 s_swizzled_tile(buf + Shared1::kNumel);

    Reg r_tile;
    Reg r_tile_swizzled;

    SIterator1 s_tiles(buf);
    SIterator2 s_swizzled_tiles(buf + Shared1::kNumel);

    for (int k = 0; k < GIterator::sc1; ++k) {
        g2s(g_tiles(k), s_tile);
        g2s_swizzled(g_tiles(k), s_swizzled_tile);
        __copy_async();
        __syncthreads();

        for (int i = 0; i < SIterator1::sc1; ++i) {
            s2r(s_tiles(i), r_tile);
            s2r(s_swizzled_tiles(i), r_tile_swizzled);
            __syncthreads();

            check_results<Reg, Element>(r_tile, r_tile_swizzled, Reg::kRows,
                                        Reg::kCols);
#ifdef DEBUG
            if (thread(0)) {
                printf("\niteration [%d, %d]\n", k, i);
                printf("r_tile:\n");
                r_tile.dump_value();

                printf("\nr_tile_swizzled:\n");
                r_tile_swizzled.dump_value();
            }
#endif
        }
    }
}

/// @brief This unit test verifies the correctness of the swizzled row-major
///        format for loading operand A in GEMM.
template <typename WarpLayout, const int kRows, const int kCols,
          const int kShmRows, const int kShmCols, const int kChunkShm>
void run_test_rowmajor() {
    static_assert(kShmRows == kRows, "kShmRows must be equal to kRows");

    using Element = __half;
    const int kThreads = tl::get_numel<WarpLayout> * 32;
    static constexpr int kWarpPerRow = tl::num_rows<WarpLayout>;

    using Global = GlobalTile<Element, tl::RowMajor<kRows, kCols>>;
    using GIterator = TileIterator<Global, TileShape<kRows, kShmCols>>;

    // for non-swizzled layout
    using Shared1 = SharedTile<Element, tl::RowMajor<kShmRows, kShmCols>,
                               false /*enable swizzled layout on shared*/>;
    using SIterator1 = TileIterator<Shared1, TileShape<kShmRows, kChunkShm>>;

    // for swizzled layout
    using Shared2 = SharedTile<Element, tl::RowMajor<kShmRows, kShmCols>,
                               true /*enable swizzled layout on shared*/>;
    using SIterator2 = TileIterator<Shared2, TileShape<kShmRows, kChunkShm>>;

    using BaseShape = traits::BaseTileShape<Element>;

    const int kSc0 = kShmRows / kWarpPerRow / BaseShape::kRows;
    const int kSc1 = kChunkShm / BaseShape::kCols;

    using Reg = RegTile<BaseTileRowMajor<Element>, tl::RowMajor<kSc0, kSc1>>;

#ifdef DEBUG
    LOG(INFO) << "GIterator: " << GIterator{} << std::endl
              << "SIterator1: " << SIterator1{} << std::endl
              << "SIterator2: " << SIterator2{} << std::endl
              << "GlobalTile Shape: [" << kRows << ", " << kCols << "]"
              << std::endl
              << "SharedTile Shape: [" << kShmRows << ", " << kShmCols << "]"
              << std::endl
              << "sc0: " << kSc0 << ", sc1: " << kSc1 << std::endl
              << "RegTile Shape: " << Reg{} << std::endl;
#endif

    using G2S1 = GlobalToSharedLoader<Shared1, WarpLayout>;
    using G2S2 = GlobalToSharedLoader<Shared2, WarpLayout>;
    using S2R = SharedToRegLoader<Reg, WarpLayout, WarpReuse::kRowReuseCont>;

    dim3 dim_grid(1, 1, 1);
    dim3 dim_block(kThreads, 1, 1);
    int shm_size = (Shared1::kNumel + Shared2::kNumel) * sizeof(Element);

    const int numel = kRows * kCols;
    __half* dA;
    CudaCheck(hipMalloc(&dA, numel * sizeof(__half)));
    const int threads = 128;
    const int blocks = CeilDiv<numel, threads>;
    init_halfs<<<blocks, threads>>>(dA, numel);

    G2S1 g2s;
    G2S2 g2s_swizzled;
    S2R s2r;

    auto test_func =
        &swizzled_copy<Element, Global, GIterator, Shared1, SIterator1, Shared2,
                       SIterator2, Reg, G2S1, G2S2, S2R>;

    // maximal statically allocated smem per block
    const int kMaxSmemPerBlock = 48 * 1024;
    if (shm_size > kMaxSmemPerBlock) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(
            test_func), hipFuncAttributeMaxDynamicSharedMemorySize, shm_size);
    }

    test_func<<<dim_grid, dim_block, shm_size>>>(dA, g2s, g2s_swizzled, s2r);
    hipDeviceSynchronize();

    CudaCheck(hipFree(dA));

    std::ostringstream ss;
    ss << "[" << kRows << ", " << kCols << ", " << kShmRows << ", " << kShmCols
       << ", " << kChunkShm << "]";
    LOG(INFO) << std::endl << ss.str() << " passed!" << std::endl;
}

/// @brief This unit test verifies the correctness of the swizzled column-major
///        format for loading operand B in GEMM.
template <typename WarpLayout, const int kRows /*K*/, const int kCols /*N*/,
          const int kShmRows, const int kShmCols, const int kChunkShm>
void run_test_colmajor() {
    using Element = __half;
    const int kThreads = tl::get_numel<WarpLayout> * 32;
    static constexpr int kWarpPerCol = tl::num_cols<WarpLayout>;

    static_assert(kShmCols == kCols, "kShmCols must be equal to kCols.");

    using Global = GlobalTile<Element, tl::ColMajor<kRows, kCols>>;
    using GIterator = TileIterator<Global, TileShape<kShmRows, kShmCols>>;

    // for non-swizzled layout
    using Shared1 = SharedTile<Element, tl::ColMajor<kShmRows, kShmCols>,
                               false /*enable swizzled layout on shared*/>;
    using SIterator1 = TileIterator<Shared1, TileShape<kChunkShm, kShmCols>>;

    // for swizzled layout
    using Shared2 = SharedTile<Element, tl::ColMajor<kShmRows, kShmCols>,
                               true /*enable swizzled layout on shared*/>;
    using SIterator2 = TileIterator<Shared2, TileShape<kChunkShm, kShmCols>>;

    using BaseShape = traits::BaseTileShape<Element>;

    const int kSc0 = kChunkShm / BaseShape::kRows;
    const int kSc1 = kShmCols / BaseShape::kCols / kWarpPerCol;

    using Reg = RegTile<BaseTileColMajor<Element>, tl::ColMajor<kSc0, kSc1>>;

#ifdef DEBUG
    LOG(INFO) << std::endl
              << "GIterator: " << GIterator{} << std::endl
              << "SIterator1: " << SIterator1{} << std::endl
              << "SIterator2: " << SIterator2{} << std::endl
              << "GlobalTile Shape: [" << kRows << ", " << kCols << "]"
              << std::endl
              << "SharedTile Shape: [" << kShmRows << ", " << kShmCols << "]"
              << std::endl
              << "sc0: " << kSc0 << ", sc1: " << kSc1 << std::endl
              << "RegTile Shape: " << Reg{} << std::endl;
#endif

    using G2S1 = GlobalToSharedLoader<Shared1, WarpLayout>;
    using G2S2 = GlobalToSharedLoader<Shared2, WarpLayout>;
    using S2R = SharedToRegLoader<Reg, WarpLayout, WarpReuse::kColReuseCont>;

    dim3 dim_grid(1, 1, 1);
    dim3 dim_block(kThreads, 1, 1);
    int shm_size = (Shared1::kNumel + Shared2::kNumel) * sizeof(Element);

    const int numel = kRows * kCols;
    __half* dA;
    CudaCheck(hipMalloc(&dA, numel * sizeof(__half)));
    const int threads = 128;
    const int blocks = CeilDiv<numel, threads>;
    init_halfs<<<blocks, threads>>>(dA, numel);

    G2S1 g2s;
    G2S2 g2s_swizzled;
    S2R s2r;

    auto test_func =
        &swizzled_copy<Element, Global, GIterator, Shared1, SIterator1, Shared2,
                       SIterator2, Reg, G2S1, G2S2, S2R>;

    // maximal statically allocated smem per block
    const int kMaxSmemPerBlock = 48 * 1024;
    if (shm_size > kMaxSmemPerBlock) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(
            test_func), hipFuncAttributeMaxDynamicSharedMemorySize, shm_size);
    }

    test_func<<<dim_grid, dim_block, shm_size>>>(dA, g2s, g2s_swizzled, s2r);
    hipDeviceSynchronize();

    CudaCheck(hipFree(dA));

    std::ostringstream ss;
    ss << "[" << kRows << ", " << kCols << ", " << kShmRows << ", " << kShmCols
       << ", " << kChunkShm << "]";
    LOG(INFO) << std::endl << ss.str() << " passed!" << std::endl;
}

template <typename Element, typename Global, typename Reg, typename Shared,
          typename Loader, typename StorerR2S, typename StorerS2G>
__global__ void swizzled_store(const Element* src, Element* dst, Loader loader,
                               StorerR2S storer1, StorerS2G storer2) {
    extern __shared__ __align__(sizeof(double)) unsigned char buf_[];
    auto* buf = reinterpret_cast<Element*>(buf_);

    Global g_src_tile(src);
    Reg r_tile;

    Shared s_tile(buf);
    Global g_dst_tile(dst);

    loader(g_src_tile, r_tile);
    __syncthreads();

    storer1(r_tile, s_tile);
    __syncthreads();

    storer2(s_tile, g_dst_tile);
    __syncthreads();

#ifdef DEBUG
    if (thread0()) {
        printf("\nreg tile:\n");
        r_tile.dump_value();

        printf("\nshared tile:\n");
        s_tile.dump_value();

        printf("\nglobal tile:\n");
        g_dst_tile.dump_value();
    }
#endif
}

template <typename Element, typename WarpLayout, const int kRows,
          const int kCols, const bool kSwizzled>
void test_row_major_store() {
    using BaseShape = traits::BaseTileShape<Element>;

    const int kThreads = tl::get_numel<WarpLayout> * 32;

    // define tiles
    using Global = GlobalTile<Element, tl::RowMajor<kRows, kCols>>;
    static constexpr int kRowRepeats =
        kRows / tl::num_rows<WarpLayout> / BaseShape::kTileSize;
    static constexpr int kColRepeats =
        kCols / tl::num_cols<WarpLayout> / BaseShape::kTileSize;

    using Reg = RegTile<BaseTileRowMajor<Element>,
                        tl::RowMajor<kRowRepeats, kColRepeats>>;
    using Shared = SharedTile<Element, tl::RowMajor<kRows, kCols>, kSwizzled>;

    // define loader and storer
    using Loader = GlobalToRegLoader<Reg, WarpLayout, copy::WarpReuse::kCont>;
    using StorerR2S = RegToSharedStorer<Reg, WarpLayout>;
    using StorerS2G = SharedToGlobalStorer<Shared, WarpLayout>;

    Loader loader;
    StorerR2S storer1;
    StorerS2G storer2;

    int numel = kRows * kCols;
    thrust::host_vector<Element> h_src(numel);
    for (int i = 0; i < h_src.size(); ++i) {
        h_src[i] = static_cast<Element>(i);
    }
    thrust::device_vector<Element> d_src = h_src;

    thrust::device_vector<Element> d_dst(numel);
    thrust::fill(d_dst.begin(), d_dst.end(), static_cast<Element>(0.));

    auto test_func = &swizzled_store<Element, Global, Reg, Shared, Loader,
                                     StorerR2S, StorerS2G>;

    dim3 dim_grid(1, 1, 1);
    dim3 dim_block(kThreads, 1, 1);
    int shm_size = Shared::kNumel * sizeof(Element);

    test_func<<<dim_grid, dim_block, shm_size>>>(
        thrust::raw_pointer_cast(d_src.data()),
        thrust::raw_pointer_cast(d_dst.data()), loader, storer1, storer2);
    hipDeviceSynchronize();

    thrust::host_vector<Element> h_dst = d_dst;

    assert_equal(thrust::raw_pointer_cast(h_src.data()),
                 thrust::raw_pointer_cast(h_dst.data()), numel, 1e-4);
};

template <typename Element, typename WarpLayout, const int kRows,
          const int kCols, const bool kSwizzled>
void test_col_major_store() {
    using BaseShape = traits::BaseTileShape<Element>;
    const int kThreads = tl::get_numel<WarpLayout> * 32;

    // define tiles
    using Global = GlobalTile<Element, tl::ColMajor<kRows, kCols>>;
    static constexpr int kRowRepeats =
        kRows / tl::num_rows<WarpLayout> / BaseShape::kTileSize;
    static constexpr int kColRepeats =
        kCols / tl::num_cols<WarpLayout> / BaseShape::kTileSize;
    using Reg = RegTile<BaseTileColMajor<Element>,
                        tl::ColMajor<kRowRepeats, kColRepeats>>;
    using Shared = SharedTile<Element, tl::ColMajor<kRows, kCols>, kSwizzled>;

    // define loader and storer
    using Loader = GlobalToRegLoader<Reg, WarpLayout, copy::WarpReuse::kCont>;
    using StorerR2S = RegToSharedStorer<Reg, WarpLayout>;
    using StorerS2G = SharedToGlobalStorer<Shared, WarpLayout>;

    Loader loader;
    StorerR2S storer1;
    StorerS2G storer2;

    int numel = kRows * kCols;
    thrust::host_vector<Element> h_src(numel);
    for (int i = 0; i < h_src.size(); ++i) {
        h_src[i] = static_cast<Element>(i);
    }
    thrust::device_vector<Element> d_src = h_src;

    thrust::device_vector<Element> d_dst(numel);
    thrust::fill(d_dst.begin(), d_dst.end(), static_cast<Element>(0.));

    auto test_func = &swizzled_store<Element, Global, Reg, Shared, Loader,
                                     StorerR2S, StorerS2G>;

    dim3 dim_grid(1, 1, 1);
    dim3 dim_block(kThreads, 1, 1);
    int shm_size = Shared::kNumel * sizeof(Element);

    test_func<<<dim_grid, dim_block, shm_size>>>(
        thrust::raw_pointer_cast(d_src.data()),
        thrust::raw_pointer_cast(d_dst.data()), loader, storer1, storer2);
    hipDeviceSynchronize();

    thrust::host_vector<Element> h_dst = d_dst;

    assert_equal(thrust::raw_pointer_cast(h_src.data()),
                 thrust::raw_pointer_cast(h_dst.data()), numel, 1e-4);
};
}  // namespace

TEST(TestSwizzledLayout, test_load_row_major) {
    run_test_rowmajor<tl::RowMajor<1, 2>, 16, 64, 16, 32, 32>();
    run_test_rowmajor<tl::RowMajor<1, 2>, 16, 128, 16, 64, 32>();
    run_test_rowmajor<tl::RowMajor<1, 2>, 32, 32, 32, 32, 16>();

    run_test_rowmajor<tl::RowMajor<2, 2>, 32, 32, 32, 32, 16>();
    run_test_rowmajor<tl::RowMajor<2, 2>, 32, 32, 32, 32, 32>();
    run_test_rowmajor<tl::RowMajor<2, 2>, 128, 256, 128, 128, 64>();

    run_test_rowmajor<tl::RowMajor<2, 1>, 32, 64, 32, 32, 32>();
    run_test_rowmajor<tl::RowMajor<2, 1>, 32, 128, 32, 64, 32>();
    run_test_rowmajor<tl::RowMajor<2, 1>, 64, 256, 64, 128, 64>();
    run_test_rowmajor<tl::RowMajor<4, 1>, 64, 64, 64, 64, 32>();
    run_test_rowmajor<tl::RowMajor<4, 1>, 64, 128, 64, 64, 64>();
    run_test_rowmajor<tl::RowMajor<4, 1>, 128, 64, 128, 64, 64>();
    run_test_rowmajor<tl::RowMajor<4, 1>, 64, 64, 64, 64, 64>();
    run_test_rowmajor<tl::RowMajor<4, 1>, 64, 128, 64, 128, 128>();
    run_test_rowmajor<tl::RowMajor<4, 1>, 64, 256, 64, 128, 128>();
    run_test_rowmajor<tl::RowMajor<8, 1>, 128, 512, 128, 256, 128>();
}

TEST(TestSwizzledLayout, test_load_col_major) {
    run_test_colmajor<tl::RowMajor<1, 1>, 16 /*K*/, 16 /*N*/, 16, 16, 16>();
    run_test_colmajor<tl::RowMajor<1, 1>, 64 /*K*/, 64 /*N*/, 32, 64, 16>();
    run_test_colmajor<tl::RowMajor<1, 2>, 128 /*K*/, 32 /*N*/, 64, 32, 32>();
    run_test_colmajor<tl::RowMajor<2, 1>, 256 /*K*/, 64 /*N*/, 128, 64, 32>();
    run_test_colmajor<tl::RowMajor<2, 2>, 256 /*K*/, 128 /*N*/, 64, 128, 32>();
    run_test_colmajor<tl::RowMajor<4, 1>, 128 /*K*/, 64 /*N*/, 64, 64, 64>();
}

TEST(TestNonSwizzledStore, test_row_major) {
    static constexpr int kSwizzled = false;
    test_row_major_store<float, tl::RowMajor<1, 1>, 16, 16, kSwizzled>();
    test_row_major_store<float, tl::RowMajor<2, 1>, 64, 32, kSwizzled>();
    test_row_major_store<float, tl::RowMajor<1, 2>, 128, 64, kSwizzled>();
    test_row_major_store<float, tl::RowMajor<2, 2>, 64, 64, kSwizzled>();

    test_row_major_store<__half, tl::RowMajor<1, 1>, 16, 16, kSwizzled>();
    test_row_major_store<__half, tl::RowMajor<2, 1>, 64, 32, kSwizzled>();
    test_row_major_store<__half, tl::RowMajor<1, 2>, 128, 64, kSwizzled>();
    test_row_major_store<__half, tl::RowMajor<2, 2>, 64, 64, kSwizzled>();
}

TEST(TestSwizzledStored, test_row_major) {
    static constexpr int kSwizzled = true;
    // bank conflict free
    test_row_major_store<float, tl::RowMajor<1, 1>, 16, 16, kSwizzled>();
    // bank conflict free
    test_row_major_store<float, tl::RowMajor<1, 1>, 16, 48, kSwizzled>();
    // bank conflict free
    test_row_major_store<float, tl::RowMajor<2, 1>, 32, 48, kSwizzled>();

    // FIXME(haruhi): below test cases have bank conflicts. In the current
    // implementation, a single `BaseTile` store/load shared memory will cause 8
    // bank conflicts.

    // This test case has 32 bank conflicts in total
    test_row_major_store<float, tl::RowMajor<1, 1>, 16, 32, kSwizzled>();
    // This test case has 128 bank conflicts in total
    test_row_major_store<float, tl::RowMajor<2, 1>, 64, 32, kSwizzled>();
    // This test case has 512 bank conflicts in total
    test_row_major_store<float, tl::RowMajor<1, 2>, 128, 64, kSwizzled>();
    // This test case has 256 bank conflicts in total
    test_row_major_store<float, tl::RowMajor<2, 2>, 64, 64, kSwizzled>();
}

TEST(TestNonSwizzledStored, test_col_major) {
    static constexpr int kSwizzled = false;
    test_col_major_store<__half, tl::RowMajor<1, 1>, 16, 16, kSwizzled>();
    test_row_major_store<__half, tl::RowMajor<2, 1>, 64, 32, kSwizzled>();
    test_row_major_store<__half, tl::RowMajor<1, 2>, 128, 64, kSwizzled>();
    test_row_major_store<__half, tl::RowMajor<2, 2>, 64, 64, kSwizzled>();

    test_col_major_store<float, tl::RowMajor<1, 1>, 16, 16, kSwizzled>();
    test_col_major_store<float, tl::RowMajor<2, 1>, 64, 32, kSwizzled>();
    test_col_major_store<float, tl::RowMajor<1, 2>, 128, 64, kSwizzled>();
    test_col_major_store<float, tl::RowMajor<2, 2>, 64, 64, kSwizzled>();
}

TEST(TestSwizzledStored, test_col_major) {
    static constexpr int kSwizzled = true;
    test_col_major_store<float, tl::RowMajor<1, 1>, 16, 16, kSwizzled>();
    test_col_major_store<float, tl::RowMajor<2, 1>, 64, 32, kSwizzled>();
    test_col_major_store<float, tl::RowMajor<1, 2>, 128, 64, kSwizzled>();
    test_col_major_store<float, tl::RowMajor<2, 2>, 64, 64, kSwizzled>();
}

}  // namespace tiledcuda::testing
