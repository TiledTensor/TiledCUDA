#include "hip/hip_runtime.h"
#include "cell/compute/reduce.hpp"
#include "cell/copy/constants.hpp"
#include "cell/copy/mod.hpp"
#include "common/test_utils.hpp"
#include "types/mod.hpp"
#include "util/debug.hpp"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

namespace tiledcuda::testing {
using namespace cell;

template <typename Element, typename RegLayout, typename GlobalLayout,
          typename BaseTile, typename WarpLayout, const tl::Layout kLayout,
          const copy::WarpReuse kMode, const int kHeight, const int kWidth>
__global__ void reg_reduce(Element* src) {
    using SrcLoadTile = GlobalTile<Element, GlobalLayout>;
    using DstLoadTile = RegTile<BaseTile, RegLayout>;
    using SrcReduceTile = DstLoadTile;
    using DstReduceTile = RegTile<Element, tl::RowMajor<kHeight, 2>>;

    SrcLoadTile src_load_tile(src);
    DstLoadTile dst_load_tile;
    DstReduceTile dst_reduce_tile;

    // Load data from global memory to register file
    copy::GlobalToRegLoader<DstLoadTile, WarpLayout, kMode> loader;
    loader(src_load_tile, dst_load_tile);
    __syncthreads();

    // Execute reduce operation.
    compute::SumReduce<SrcReduceTile, kLayout> row_sum;
    row_sum(dst_load_tile, dst_reduce_tile);

    __syncthreads();
}

}  // namespace tiledcuda::testing
