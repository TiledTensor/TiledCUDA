#include "hip/hip_runtime.h"
#include "common/test_utils.hpp"
#include "types/register.hpp"
#include "types/shared.hpp"

namespace tiledcuda {
namespace testing {

using namespace cell;
using namespace cute;
namespace tl = tile_layout;

namespace {

template <typename Shared>
__device__ void init_tile(Shared& data) {
    int v = 0;
    for (int i = 0; i < Shared::kRows; ++i) {
        for (int j = 0; j < Shared::kCols; ++j) {
            data(i, j) = static_cast<typename Shared::DType>(v++);
        }
    }
}

template <typename Shared>
__global__ void test_shared_tile() {
    using DType = typename Shared::DType;

    extern __shared__ __align__(sizeof(double)) unsigned char buf_[];
    auto* buf = reinterpret_cast<typename Shared::DType*>(buf_);

    Shared s_tile(buf);
    init_tile(s_tile);

    printf("Shared tile:\n");
    s_tile.dump_value();

    using Reg = RegTile<DType, tl::RowMajor<2, 4>>;
    Reg r_tile;

    init_tile(r_tile);

    printf("Shared tile:\n");
    r_tile.dump_value();
}

}  // namespace

TEST(TestTile, test_shared_tile) {
    using Element = cutlass::half_t;

    const int rows = 16;
    const int cols = 8;

    using Shared1 = SharedTile<Element, tl::RowMajor<rows, cols>>;
    int shm_size = Shared1::kNumel * sizeof(Element);
    // DONOT change this launch config. The unittest is implemented for a single
    // thread.
    test_shared_tile<Shared1><<<1, 1, shm_size>>>();
    hipDeviceSynchronize();
}

}  // namespace testing
}  // namespace tiledcuda
