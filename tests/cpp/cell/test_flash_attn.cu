#include "hip/hip_runtime.h"
#include "cell/compute/broadcast.hpp"
#include "cell/compute/map.hpp"
#include "cell/compute/reduce.hpp"
#include "cell/copy/constants.hpp"
#include "cell/copy/mod.hpp"
#include "common/test_utils.hpp"
#include "types/mod.hpp"
#include "util/debug.hpp"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

namespace tiledcuda::testing {
using namespace cell;

/**
 * @brief Reduce/Map operation for the flash attention.
 */
template <typename Element, typename RegLayout, typename GlobalLayout,
          typename ReduceLayout, typename BaseTile, typename WarpLayout,
          const tl::Layout kLayout, const copy::WarpReuse kMode,
          const int kHeight, const int kWidth>
__global__ void flash_attn_reg_reduce(Element* src) {
    using SrcLoadTile = GlobalTile<Element, GlobalLayout>;
    using DstLoadTile = RegTile<BaseTile, RegLayout>;
    using SrcReduceTile = DstLoadTile;
    using DstReduceTile = RegTile<Element, tl::RowMajor<kHeight, 2>>;
    using SrcBroadcastTile = DstReduceTile;
    using DstBroadcastTile = SrcReduceTile;

    SrcLoadTile src_load_tile(src);
    DstLoadTile attn_block;
    DstReduceTile last_max_vec;
    DstReduceTile max_vec;
    DstReduceTile last_norm_vec;
    DstReduceTile norm_vec;
    DstBroadcastTile max_broadcast_tile;
    DstBroadcastTile norm_broadcast_tile;

    // Load data from global memory to register file
    copy::GlobalToRegLoader<DstLoadTile, WarpLayout, kMode> loader;
    loader(src_load_tile, attn_block);

    // Copy `max_vec` into `last_max_vec`
    copy::BaseTileCopy<DstReduceTile> copy_max_reg;
    copy_max_reg(max_vec, last_max_vec);
    // Copy `norm_vec` into `last_norm_vec`
    copy::BaseTileCopy<DstReduceTile> copy_norm_reg;
    copy_norm_reg(norm_vec, last_norm_vec);

    // Execute reduce operation.
    compute::MaxReduce<SrcReduceTile, kLayout> row_max;
    // accumulate onto the max_vec
    row_max(attn_block, max_vec);

    compute::Broadcast<SrcBroadcastTile, DstBroadcastTile, kLayout>
        broadcast_max;

    broadcast_max(max_vec, max_broadcast_tile);

    if (thread(0)) {
        printf("Thread 0:\n");
        max_vec.dump_value();
        max_broadcast_tile.dump_value();
        attn_block.dump_value();
    }

    // subtract max from attention -- now all <= 0.
    compute::RegTileSub<DstBroadcastTile> sub_row_max;
    sub_row_max(attn_block, max_broadcast_tile, attn_block);

    if (thread(0)) {
        printf("Thread 0:\n");
        attn_block.dump_value();
    }

    // exponentiate the block in-place.
    compute::RegTileExp<DstBroadcastTile> exp_attn;
    exp_attn(attn_block, attn_block);

    if (thread(0)) {
        printf("Thread 0:\n");
        attn_block.dump_value();
    }

    // subtract new max from old max to find the new normalization.
    compute::BaseTileSub<DstReduceTile> sub_new_max;
    sub_new_max(last_max_vec, max_vec, last_max_vec);

    // exponentiate this vector -- this is what we need to normalize by.
    compute::BaseTileExp<DstReduceTile> exp_max;
    exp_max(last_max_vec, last_max_vec);

    // and the norm vec is now normalized.
    compute::BaseTileMul<DstReduceTile> mul_norm;
    mul_norm(last_max_vec, norm_vec, norm_vec);

    // Accumulate the new attention block onto the now-rescaled norm-vec.
    // Reduce Sum + Add
    DstReduceTile sum_vec;
    compute::SumReduce<SrcReduceTile, kLayout> row_sum;
    row_sum(attn_block, sum_vec);
    compute::BaseTileAdd<DstReduceTile> add_sum;
    add_sum(sum_vec, norm_vec, norm_vec);

    // Now the attention block is correctly normalized.
    // Broadcast + Divide
    compute::Broadcast<SrcBroadcastTile, DstBroadcastTile, kLayout>
        broadcast_norm;
    broadcast_norm(norm_vec, norm_broadcast_tile);
    compute::RegTileDiv<DstBroadcastTile> div_norm;
    div_norm(attn_block, norm_broadcast_tile, attn_block);

    // Normalize the previous norm vec accorfing to the new max.
    compute::BaseTileMul<DstReduceTile> mul_norm_new;
    mul_norm_new(last_max_vec, last_norm_vec, last_norm_vec);

    // Normalize the previous norm vec according to the new norm.
    compute::BaseTileDiv<DstReduceTile> div_norm_new;
    div_norm_new(last_norm_vec, norm_vec, last_norm_vec);
}

template <typename Element, typename RegLayout, typename GlobalLayout,
          typename BaseTile, typename WarpLayout, const tl::Layout kLayout,
          const copy::WarpReuse kMode, const int kHeight, const int kWidth>
void run_row_major_reg_flash_attn() {
    int kNumel = 16 * 16 * kHeight * kWidth;
    int kWarpSize = tl::get_numel<WarpLayout>;

    using ReduceLayout = tl::RowMajor<kHeight, 2>;

    thrust::host_vector<Element> h_src(kNumel);
    for (int i = 0; i < kNumel; ++i) {
        h_src[i] = (Element)i;
    }

    thrust::device_vector<Element> d_src = h_src;

    flash_attn_reg_reduce<Element, RegLayout, GlobalLayout, ReduceLayout,
                          BaseTile, WarpLayout, kLayout, kMode, kHeight, kWidth>
        <<<1, 32 * kWarpSize>>>(thrust::raw_pointer_cast(d_src.data()));
}

TEST(TestRegBroadcast, row_major_reg_flash_attn_0) {
    const int kHeight = 1;
    const int kWidth = 1;
    using Element = float;
    using WarpLayout = tl::RowMajor<1, 1>;
    using RegLayout = tl::RowMajor<kHeight, kWidth>;

    const copy::WarpReuse kMode = copy::WarpReuse::kCont;

    using GlobalLayout = tl::RowMajor<16 * kHeight, 16 * kWidth>;

    run_row_major_reg_flash_attn<
        Element, RegLayout, GlobalLayout, BaseTileRowMajor<Element>, WarpLayout,
        tl::Layout::kRowMajor, kMode, kHeight, kWidth>();
}

}  // namespace tiledcuda::testing
