#include "hip/hip_runtime.h"
#include "cell/compute/broadcast.hpp"
#include "cell/compute/map.hpp"
#include "cell/compute/reduce.hpp"
#include "cell/copy/constants.hpp"
#include "cell/copy/mod.hpp"
#include "common/test_utils.hpp"
#include "types/mod.hpp"
#include "util/debug.hpp"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

namespace tiledcuda::testing {
using namespace cell;

template <typename Element, typename RegLayout, typename GlobalLayout,
          typename ReduceLayout, typename BaseTile, typename WarpLayout,
          const tl::Layout kLayout, const copy::WarpReuse kMode,
          const int kHeight, const int kWidth>
__global__ void flash_attn_reg_reduce(Element* src) {
    using SrcLoadTile = GlobalTile<Element, GlobalLayout>;
    using DstLoadTile = RegTile<BaseTile, RegLayout>;
    using SrcReduceTile = DstLoadTile;
    using DstReduceTile = RegTile<Element, tl::RowMajor<kHeight, 2>>;
    using SrcBroadcastTile = DstReduceTile;
    using DstBroadcastTile = SrcReduceTile;

    SrcLoadTile src_load_tile(src);
    DstLoadTile attn_block;
    DstReduceTile last_max_vec;
    DstReduceTile max_vec;
    DstBroadcastTile max_broadcast_tile;

    // Load data from global memory to register file
    copy::GlobalToRegLoader<DstLoadTile, WarpLayout, kMode> loader;
    loader(src_load_tile, attn_block);

    // Copy `max_vec` into `last_max_vec`
    copy::BaseTileCopy<DstReduceTile> copy_max_reg;
    copy_max_reg(max_vec, last_max_vec);

    // Execute reduce operation.
    compute::MaxReduce<SrcReduceTile, kLayout> row_max;
    // accumulate onto the max_vec
    row_max(attn_block, max_vec);

    compute::Broadcast<SrcBroadcastTile, DstBroadcastTile, kLayout>
        broadcast_max;

    broadcast_max(max_vec, max_broadcast_tile);

    if (thread(0)) {
        printf("Thread 0:\n");
        max_vec.dump_value();
        max_broadcast_tile.dump_value();
        attn_block.dump_value();
    }

    // subtract max from attention -- now all <= 0.
    compute::RegTileSub<DstBroadcastTile> sub_row_max;
    sub_row_max(attn_block, max_broadcast_tile, attn_block);

    if (thread(0)) {
        printf("Thread 0:\n");
        attn_block.dump_value();
    }

    // exponentiate the block in-place.
    compute::RegTileExp<DstBroadcastTile> exp_attn;
    exp_attn(attn_block, attn_block);

    if (thread(0)) {
        printf("Thread 0:\n");
        attn_block.dump_value();
    }
}

template <typename Element, typename RegLayout, typename GlobalLayout,
          typename BaseTile, typename WarpLayout, const tl::Layout kLayout,
          const copy::WarpReuse kMode, const int kHeight, const int kWidth>
void run_row_major_reg_flash_attn() {
    int kNumel = 16 * 16 * kHeight * kWidth;
    int kWarpSize = tl::get_numel<WarpLayout>;

    using ReduceLayout = tl::RowMajor<kHeight, 2>;

    thrust::host_vector<Element> h_src(kNumel);
    for (int i = 0; i < kNumel; ++i) {
        h_src[i] = (Element)i;
    }

    thrust::device_vector<Element> d_src = h_src;

    flash_attn_reg_reduce<Element, RegLayout, GlobalLayout, ReduceLayout,
                          BaseTile, WarpLayout, kLayout, kMode, kHeight, kWidth>
        <<<1, 32 * kWarpSize>>>(thrust::raw_pointer_cast(d_src.data()));
}

TEST(TestRegBroadcast, row_major_reg_flash_attn_0) {
    const int kHeight = 1;
    const int kWidth = 1;
    using Element = float;
    using WarpLayout = tl::RowMajor<1, 1>;
    using RegLayout = tl::RowMajor<kHeight, kWidth>;

    const copy::WarpReuse kMode = copy::WarpReuse::kCont;

    using GlobalLayout = tl::RowMajor<16 * kHeight, 16 * kWidth>;

    run_row_major_reg_flash_attn<
        Element, RegLayout, GlobalLayout, BaseTileRowMajor<Element>, WarpLayout,
        tl::Layout::kRowMajor, kMode, kHeight, kWidth>();
}

}  // namespace tiledcuda::testing
