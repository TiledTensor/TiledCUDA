#include "hip/hip_runtime.h"
#include "cell/mod.hpp"
#include "common/test_utils.hpp"
#include "cuda_utils.hpp"
#include "util/debug.hpp"

#include <hipblas.h>
#include <glog/logging.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

namespace tiledcuda {

using namespace cell;
using namespace cell::copy;
namespace tl = tile_layout;

using namespace cute;

namespace {

float rand_float(float a = 1e-3, float b = 1) {
    float random = ((float)rand()) / (float)RAND_MAX;
    float diff = b - a;
    float r = random * diff;
    return a + r;
}

void check_correctness(const half* hc1, const float* hc2, const float* hc3,
                       int numel) {
    printf("cublas:\n");
    for (int i = 0; i < numel / 16; ++i) {
        printf("%.3f, ", __half2float(hc1[i]));
        if (i && (i + 1) % 8 == 0) printf("\n");
        // ASSERT_NEAR(hc1[i], hc2[i], 1e-3);
    }
    printf("\nnaive:\n");
    for (int i = 0; i < numel / 16; ++i) {
        printf("%.3f, ", hc2[i]);
        if (i && (i + 1) % 8 == 0) printf("\n");
        // ASSERT_NEAR(hc1[i], hc2[i], 1e-3);
    }
    printf("\n");

    printf("ours:\n");
    for (int i = 0; i < numel / 16; ++i) {
        printf("%.3f, ", hc3[i]);
        if (i && (i + 1) % 8 == 0) printf("\n");
        // ASSERT_NEAR(hc1[i], hc2[i], 1e-3);
    }
    printf("\n");
}

// In this implementation, A and C are interpreted as being laid out in
// row-major, and B is interpreted as being laid out in column-major.
void naive_gemm(int kM, int kN, int kK,  //
                const __half* A, const __half* B, float* C) {
    for (int i = 0; i < kM; ++i) {
        for (int j = 0; j < kN; ++j) {
            float s = 0.;
            for (int k = 0; k < kK; ++k) {
                s += __half2float(A[i * kK + k]) * __half2float(B[j * kK + k]);
            }
            C[i * kN + j] = s;
        }
    }
}

// @brief: This implementation interprets A and C as being laid out in row-major
//        order, while B is laid out in column-major order.
void cublas_hgemm(int m, int n, int k, const __half* A, const __half* B,
                  __half* C, int lda, int ldb, int ldc) {
    __half alf = 1.;
    __half bet = 0.;

    hipblasHandle_t handle;
    CublasCheck(hipblasCreate(&handle));
    CublasCheck(hipblasHgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, &alf, A,
                            lda, B, ldb, &bet, C, ldc));
    CublasCheck(hipblasDestroy(handle));
}

template <typename Element, typename ElementAcc, typename WarpLayout,
          const int M, const int N, const int K>
struct TestTraits {
    static const int kThreads = tl::get_numel<WarpLayout> * 32;

    // for global to shared memory copy using CuTe
    using LoadSharedA = traits::G2S2DCopyTraits<Element, M, K, M, K, kThreads,
                                                false /*use swizzle*/>;
    using LoadSharedB = traits::G2S2DCopyTraits<Element, N, K, N, K, kThreads,
                                                false /*use swizzle*/>;
    // transfer operand C from shared memory to global memory
    using StoreSharedC =
        traits::S2G2DCopyTraits<ElementAcc, M, N, M, N, kThreads,
                                false /*use swizzle*/>;

    // ============= shared to register loader =================
    using SharedA = SharedTile<Element, tl::RowMajor<M, K>>;
    // [64, 128] is chunked by [32, 32], strip counts = [64/64, 128/32] = [1, 4]
    using TileIteratorA = SharedTileIterator<SharedA, TileShape<32, 32>>;
    using RegA = RegTile<Element, tl::RowMajor<2, 8>>;
    using LoadRegA =
        SharedToRegLoader<RegA, WarpLayout, WarpReuse::RowReuseCont,
                          CopyInst::LoadMat>;

    // A row-major Tile with a shape of [N, K] is equivalent to a column-major
    // Tile with a shape of [K, N]
    using SharedB = SharedTile<Element, tl::ColMajor<K, N>>;  // 64, 128
    using RegB = RegTile<Element, tl::RowMajor<2, 8>>;
    // [64, 128] is chunked by [32, 32], strip counts = [64/64, 128/32] = [1, 4]
    using TileIteratorB = SharedTileIterator<SharedB, TileShape<32, 32>>;
    using LoadRegB =
        SharedToRegLoader<RegB, WarpLayout, WarpReuse::ColReuseCont,
                          CopyInst::LoadMat>;

    static_assert(TileIteratorA::sc1 == TileIteratorB::sc1,
                  "dimension mismatch!");

    // ============= register to shared storer =================
    using SharedC = SharedTile<ElementAcc, tl::RowMajor<M, N>>;  // 64, 64
    using RegC = RegTile<ElementAcc, tl::RowMajor<2, 4>>;

    using StoreRegC =
        RegToSharedStorer<RegC, WarpLayout, RegLayout::WMMA_m16n16k16,
                          CopyInst::LoadS32>;
};

template <typename Element, typename ElementAcc,                              //
          typename LoadSharedA, typename LoadSharedB, typename StoreSharedC,  //
          typename TileIteratorA, typename RegA, typename LoadRegA,
          typename TileIteratorB, typename RegB, typename LoadRegB,
          typename SharedC, typename RegC, typename StoreRegC>
__global__ void test_wmma(const Element* ga, const Element* gb, ElementAcc* gc,
                          LoadRegA& load_rA, LoadRegB& load_rB,
                          StoreRegC& store_rC) {
    extern __shared__ __align__(sizeof(double)) unsigned char buf_[];
    auto* shared_a = reinterpret_cast<Element*>(buf_);
    auto* shared_b = shared_a + TileIteratorA::Tile::kNumel;
    auto* shared_c = reinterpret_cast<ElementAcc*>(buf_);
    SharedC sC(shared_c);

    // transfer tiles from global to shared memory
    copy_2d_tile_g2s(ga, shared_a, typename LoadSharedA::SrcLayout{},
                     typename LoadSharedA::DstLayout{},
                     typename LoadSharedA::TiledCopy{});
    copy_2d_tile_g2s(gb, shared_b, typename LoadSharedB::SrcLayout{},
                     typename LoadSharedB::DstLayout{},
                     typename LoadSharedB::TiledCopy{});
    __copy_async();
    __syncthreads();

    TileIteratorA sAs(shared_a);
    TileIteratorB sBs(shared_b);

    RegA rA;
    RegB rB;
    RegC acc;

    for (int k = 0; k < TileIteratorA::sc1; ++k) {
        auto sA = sAs(k);
        auto sB = sBs(k);

        load_rA(sA, rA);
        load_rB(sB, rB);

        compute::gemm_(rA, rB, acc);
    }
    __syncthreads();

    store_rC(acc, sC);
    __syncthreads();

    copy_2d_tile_s2g(shared_c, gc, typename StoreSharedC::SrcLayout{},
                     typename StoreSharedC::DstLayout{},
                     typename StoreSharedC::TiledCopy{});
}

}  // namespace

namespace testing {

// #define DEBUG

template <const int M, const int N, const int K, typename WarpLayout>
void run_test_gemm() {
    /// unittest for register-level gemm by calling into wmma PTX
    using Element = cutlass::half_t;
    using ElementAcc = float;

    // initialize data
    thrust::host_vector<Element> h_a(M * K);
    for (int i = 0; i < h_a.size(); ++i) {
#ifdef DEBUG
        h_a[i] = static_cast<Element>(i % 2048);
#else
        h_a[i] = static_cast<Element>(rand_float());
#endif
    }

    thrust::host_vector<Element> h_b(K * N);
    for (int i = 0; i < h_b.size(); ++i) {
#ifdef DEBUG
        h_b[i] = static_cast<Element>(i % 2048);
#else
        h_b[i] = static_cast<Element>(rand_float());
#endif
    }
    thrust::host_vector<ElementAcc> h_c(M * N);
    thrust::fill(h_c.begin(), h_c.end(), 0.);

    thrust::host_vector<ElementAcc> h_naive(M * N);
    thrust::fill(h_c.begin(), h_c.end(), 0.);

    naive_gemm(M, N, K, reinterpret_cast<const __half*>(h_a.data()),
               reinterpret_cast<const __half*>(h_b.data()), h_naive.data());

    thrust::device_vector<Element> d_a = h_a;
    thrust::device_vector<Element> d_b = h_b;
    thrust::device_vector<ElementAcc> d_c = h_c;

    /// define the configuration of the test
    using config = TestTraits<Element, ElementAcc, WarpLayout, M, N, K>;

    LOG(INFO) << "kThreads: " << config::kThreads << std::endl;
    LOG(INFO) << "TileIteratorA: [" << config::TileIteratorA::Tile::kRows
              << ", " << config::TileIteratorA::Tile::kCols
              << "]; numel = " << config::TileIteratorA::Tile::kNumel
              << ", sc0 = " << config::TileIteratorA::sc0
              << ", sc1 = " << config::TileIteratorA::sc1 << std::endl;
    LOG(INFO) << "TileIteratorB: [" << config::TileIteratorB::Tile::kRows
              << ", " << config::TileIteratorB::Tile::kCols
              << "]; numel = " << config::TileIteratorB::Tile::kNumel
              << ", sc0 = " << config::TileIteratorB::sc0
              << ", sc1 = " << config::TileIteratorB::sc1 << std::endl
              << std::endl;

    dim3 dim_grid(1, 1, 1);
    dim3 dim_block(config::kThreads, 1, 1);
    int size_ab = (M + N) * K * sizeof(Element);
    int size_c = M * N * sizeof(ElementAcc);
    int shm_size = size_ab > size_c ? size_ab : size_c;

    typename config::LoadRegA load_rA;
    typename config::LoadRegB load_rB;
    typename config::StoreRegC store_rC;

    // TODO: Refine this code; there are too many template parameters, making it
    // messy.
    test_wmma<Element, ElementAcc, typename config::LoadSharedA,
              typename config::LoadSharedB, typename config::StoreSharedC,
              typename config::TileIteratorA, typename config::RegA,
              typename config::LoadRegA, typename config::TileIteratorB,
              typename config::RegB, typename config::LoadRegB,
              typename config::SharedC, typename config::RegC,
              typename config::StoreRegC><<<dim_grid, dim_block, shm_size>>>(
        thrust::raw_pointer_cast(d_a.data()),
        thrust::raw_pointer_cast(d_b.data()),
        thrust::raw_pointer_cast(d_c.data()), load_rA, load_rB, store_rC);
    hipDeviceSynchronize();
    h_c = d_c;

    /// unittest for correctness, take cublas as the ground-truth
    thrust::device_vector<__half> d_cublas(M * N);
    thrust::fill(d_cublas.begin(), d_cublas.end(), 0.);

    // Matrix A has a row-major layout with dimensions [M, K],
    // Matrix B has a column-major layout with dimensions [K, N],
    // and Matrix C has a row-major layout with dimensions [M, N].
    //
    // This is equivalent to the following:
    // Matrix A has a column-major layout with dimensions [K, M],
    // Matrix B has a column-major layout with dimensions [K, N],
    // and Matrix C has a column-major layout with dimensions [N, M].
    // cuBlas is a Fortran-style(column-major) BLAS library,
    // then we compute: C = B^T @ A
    //             [N, M] = [N, K] @ [K, M]
    cublas_hgemm(
        N, M, K,
        reinterpret_cast<const __half*>(thrust::raw_pointer_cast(d_b.data())),
        reinterpret_cast<const __half*>(thrust::raw_pointer_cast(d_a.data())),
        reinterpret_cast<__half*>(thrust::raw_pointer_cast(d_cublas.data())),
        K /*lda*/, K /*ldb*/, N /*ldc*/);

    thrust::host_vector<__half> h_groundtruth = d_cublas;

#ifndef DEBUG
    check_correctness(thrust::raw_pointer_cast(h_groundtruth.data()),
                      thrust::raw_pointer_cast(h_naive.data()),
                      thrust::raw_pointer_cast(h_c.data()), M * N);
#endif
}

TEST(TestWmma, TestGemm) {
    // M, N, K is this test is for shared memory tile
    run_test_gemm<32, 32, 32, tl::RowMajor<2, 2>>();
}

}  // namespace testing
}  // namespace tiledcuda
